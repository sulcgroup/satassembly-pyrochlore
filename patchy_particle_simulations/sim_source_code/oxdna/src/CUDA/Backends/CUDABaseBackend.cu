#include "hip/hip_runtime.h"
/*
 * CUDABaseBackend.cpp
 *
 *  Created on: 25/nov/2010
 *      Author: lorenzo
 */

#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include "CUDABaseBackend.h"
#include "../Lists/CUDAListFactory.h"
#include "../Interactions/CUDAInteractionFactory.h"
#include "../../Utilities/oxDNAException.h"
#include "../../Utilities/ConfigInfo.h"

using namespace std;

// these pragma instructions remove a few nvcc warnings
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wvla"

template<typename number, typename number4>
CUDABaseBackend<number, number4>::CUDABaseBackend() : _device_number(0), _sort_every(0) {
	_particles_kernel_cfg.blocks = dim3(1, 1, 1);
	_particles_kernel_cfg.threads_per_block = 0;
	_particles_kernel_cfg.shared_mem = 0;

	_device_number = -1;
	_sqr_verlet_skin = 0.f;

	_cuda_lists = NULL;
	_cuda_interaction = NULL;
	_d_poss = NULL;
	_d_bonds = NULL;
	_d_orientations = NULL;
	_d_list_poss = NULL;
	_d_are_lists_old = NULL;
	_d_hindex = NULL;
	_d_sorted_hindex = NULL;
	_d_inv_sorted_hindex = NULL;
	_d_buff_poss = NULL;
	_d_buff_bonds = NULL;
	_d_buff_orientations = NULL;
	_d_cuda_box = NULL;
	_h_poss = NULL;
	_h_orientations = NULL;
	_h_bonds = NULL;

	_vec_size = 0;
	_bonds_size = 0;
	_orient_size = 0;
}

template<typename number, typename number4>
CUDABaseBackend<number, number4>::~CUDABaseBackend() {
	if (_cuda_lists != NULL) {
		_cuda_lists->clean();
		delete _cuda_lists;
	}
	if (_cuda_interaction != NULL) delete _cuda_interaction;

	if (_d_poss != NULL){
		CUDA_SAFE_CALL( hipFree(_d_poss) );
		CUDA_SAFE_CALL( hipFree(_d_bonds) );
		CUDA_SAFE_CALL( hipFree(_d_orientations) );
		CUDA_SAFE_CALL( hipFree(_d_list_poss) );
		CUDA_SAFE_CALL( hipHostFree(_d_are_lists_old) );
	}

	if(_sort_every > 0) {
		if (_d_hindex != NULL){
			CUDA_SAFE_CALL( hipFree(_d_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_sorted_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_inv_sorted_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_buff_poss) );
			CUDA_SAFE_CALL( hipFree(_d_buff_bonds) );
			CUDA_SAFE_CALL( hipFree(_d_buff_orientations) );
		}
	}

	if (_h_poss != NULL) delete[] _h_poss;
	if (_h_orientations != NULL) delete[] _h_orientations;
	if (_h_bonds != NULL) delete[] _h_bonds;
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_host_to_gpu() {
	CUDA_SAFE_CALL( hipMemcpy(_d_poss, _h_poss, _vec_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(_d_bonds, _h_bonds, _bonds_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(_d_orientations, _h_orientations, _orient_size, hipMemcpyHostToDevice) );

	_h_cuda_box.set_CUDA_from_CPU(CONFIG_INFO->box);
	CUDA_SAFE_CALL( hipMemcpy(_d_cuda_box, &_h_cuda_box, sizeof(CUDABox<number, number4>), hipMemcpyHostToDevice) );
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_gpu_to_host() {
	CUDA_SAFE_CALL( hipMemcpy(_h_poss, _d_poss, _vec_size, hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(_h_bonds, _d_bonds, _bonds_size, hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(_h_orientations, _d_orientations, _orient_size, hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL( hipMemcpy(&_h_cuda_box, _d_cuda_box, sizeof(CUDABox<number, number4>), hipMemcpyDeviceToHost) );
	_h_cuda_box.set_CPU_from_CUDA(CONFIG_INFO->box);
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::get_settings(input_file &inp) {
	if(getInputInt(&inp, "CUDA_device", &_device_number, 0) == KEY_NOT_FOUND) {
		OX_LOG(Logger::LOG_INFO, "CUDA device not specified");
		_device_number = -1;
	}
	else OX_LOG(Logger::LOG_INFO, "Using CUDA device %d", _device_number);

	if(getInputInt(&inp, "CUDA_sort_every", &_sort_every, 0) == KEY_NOT_FOUND)
		OX_LOG(Logger::LOG_INFO, "CUDA sort_every not specified, using 0");

	getInputInt(&inp, "threads_per_block", &_particles_kernel_cfg.threads_per_block, 0);

	float verlet_skin;
	if(getInputFloat(&inp, "verlet_skin", &verlet_skin, 0) == KEY_FOUND) _sqr_verlet_skin = SQR(verlet_skin);

	_cuda_interaction = CUDAInteractionFactory::make_interaction<number, number4>(inp);
	_cuda_interaction->get_settings(inp);
	_cuda_interaction->get_cuda_settings(inp);

	_cuda_lists = CUDAListFactory::make_list<number, number4>(inp);
	_cuda_lists->get_settings(inp);

	string reload_from;
	if(getInputString(&inp, "reload_from", reload_from, 0) == KEY_FOUND) throw oxDNAException("The CUDA backend does not support reloading checkpoints, owing to its intrisincally stochastic nature");
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_choose_device () {
	OX_LOG(Logger::LOG_INFO, "Choosing device automatically");

	int ndev = -1, trydev = 0;
	hipDeviceProp_t tryprop;

	hipGetDeviceCount (&ndev);
	OX_LOG(Logger::LOG_INFO, "Computer has %i devices", ndev);
	while (trydev < ndev) {
		OX_LOG(Logger::LOG_INFO, " - Trying device %i", trydev);
		tryprop = get_device_prop (trydev);
		OX_LOG(Logger::LOG_INFO, " -- device %i has properties %i.%i", trydev, tryprop.major, tryprop.minor);
		if (tryprop.major < 2 && tryprop.minor <= 2) {
			OX_LOG(Logger::LOG_INFO, " -- Device properties are not good. Skipping it", trydev);
			trydev ++;
			continue;
		}
		set_device (trydev);
		int *dummyptr = NULL;
		hipError_t ggg = GpuUtils::LR_cudaMalloc<int> (&dummyptr, (size_t)sizeof(int));
		if(ggg == hipSuccess) {
			OX_LOG(Logger::LOG_INFO, " -- using device %i", trydev);
			hipFree (dummyptr);
			break;
		}
		else {
			OX_LOG(Logger::LOG_INFO, " -- device %i not available ...", trydev);
		}
		trydev++;
	}

	if (trydev == ndev) throw oxDNAException("No suitable devices available");

	OX_LOG(Logger::LOG_INFO, " --- Running on device %i", trydev);
	_device_prop = get_device_prop(trydev);
	_device_number = trydev;
	// gpu device chosen
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::init_cuda() {
	if(_device_number < 0) _choose_device();
	set_device(_device_number);
	_device_prop = get_device_prop(_device_number);

	CUDA_SAFE_CALL( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );

	number box_side = CONFIG_INFO->box->box_sides().x;
	int N = *CONFIG_INFO->N;
	_h_cuda_box.set_CUDA_from_CPU(CONFIG_INFO->box);

	_cuda_interaction->cuda_init(box_side, N);

	_vec_size = sizeof(number4) * N;
	_orient_size = sizeof(GPU_quat<number>) * N;
	_bonds_size = sizeof(LR_bonds) * N;

	// GPU memory allocations
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_poss, _vec_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<LR_bonds>(&_d_bonds, _bonds_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number>  >(&_d_orientations, _orient_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_list_poss, _vec_size) );
	// the CUDA_SAFE_CALL macro does not support templates with more than one argument
	GpuUtils::LR_cudaMalloc<CUDABox<number, number4> >(&_d_cuda_box, sizeof(CUDABox<number, number4>));
	CUDA_SAFE_CALL( hipHostMalloc(&_d_are_lists_old, sizeof(bool), hipHostMallocDefault) );

	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_list_poss, _vec_size) );

	CUDA_SAFE_CALL( hipMemset(_d_list_poss, 0, _vec_size) );

	// CPU memory allocations
	_h_poss = new number4[N];
	_h_orientations = new GPU_quat<number>[N];
	_h_bonds = new LR_bonds[N];

	// setup kernels' configurations
	_init_CUDA_kernel_cfgs();
	_cuda_lists->init(N, _cuda_interaction->get_cuda_rcut(), &_h_cuda_box, _d_cuda_box);

	if(_sort_every > 0) {
		int uns = 0;

		// fixed value for depth (8): changing this value does not significantly affect performances
		init_hilb_symbols(N, uns, 8, (float) box_side);

		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_sorted_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_inv_sorted_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_buff_poss, _vec_size) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<LR_bonds>(&_d_buff_bonds, _bonds_size) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number>  >(&_d_buff_orientations, _orient_size) );

		reset_sorted_hindex
			<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
			(_d_sorted_hindex);
	}
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_init_CUDA_kernel_cfgs() {
	if(_particles_kernel_cfg.threads_per_block == 0) {
		_particles_kernel_cfg.threads_per_block = 2*_device_prop.warpSize;
		OX_LOG(Logger::LOG_INFO, "threads_per_block was not specified or set to 0. The default value (%d) will be used", 2*_device_prop.warpSize);
	}

	int N = *CONFIG_INFO->N;
	_particles_kernel_cfg.blocks.x = N / _particles_kernel_cfg.threads_per_block + ((N % _particles_kernel_cfg.threads_per_block == 0) ? 0 : 1);
	if(_particles_kernel_cfg.blocks.x == 0) _particles_kernel_cfg.blocks.x = 1;
	_particles_kernel_cfg.blocks.y = _particles_kernel_cfg.blocks.z = 1;

	_cuda_interaction->set_launch_cfg(_particles_kernel_cfg);

	OX_DEBUG("Particle kernel cfg: threads_per_block = %d, blocks = (%d, %d, %d)", _particles_kernel_cfg.threads_per_block,
			_particles_kernel_cfg.blocks.x, _particles_kernel_cfg.blocks.y, _particles_kernel_cfg.blocks.z);
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_sort_index() {
	reset_sorted_hindex
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_sorted_hindex);
	CUT_CHECK_ERROR("reset_sorted_hindex error");

	hilbert_curve<number4>
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_poss, _d_hindex);
	CUT_CHECK_ERROR("hilbert_curve error");

	thrust::device_ptr<int> _d_hindex_p(_d_hindex);
	thrust::device_ptr<int> _d_sorted_hindex_p(_d_sorted_hindex);
	// sort d_sorted_hindex by using d_hindex
	thrust::sort_by_key(_d_hindex_p, _d_hindex_p + *CONFIG_INFO->N, _d_sorted_hindex_p);
	get_inverted_sorted_hindex
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_sorted_hindex, _d_inv_sorted_hindex);
}

// template instantiations
template class CUDABaseBackend<float, float4>;
template class CUDABaseBackend<double, LR_double4>;

#pragma GCC diagnostic pop

