#include "hip/hip_runtime.h"
/*
 * CUDASimpleVerletList.cu
 *
 *  Created on: 29/set/2010
 *      Author: lorenzo
 */

#include "CUDASimpleVerletList.h"
#include "CUDA_simple_verlet.cuh"
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include "../../Utilities/oxDNAException.h"

template<typename number, typename number4>
CUDASimpleVerletList<number, number4>::CUDASimpleVerletList() : _max_density_multiplier(1) {
	_auto_optimisation = true;
	_cells_kernel_cfg.threads_per_block = 0;
	this->_use_edge = false;
	_N_cells = _old_N_cells = -1;

	_d_cells = _d_counters_cells = _d_matrix_neighs = _d_number_neighs = NULL;
}

template<typename number, typename number4>
CUDASimpleVerletList<number, number4>::~CUDASimpleVerletList() {
	_d_matrix_neighs = NULL;
	_d_number_neighs = NULL;
}

template<typename number, typename number4>
void CUDASimpleVerletList<number, number4>::clean() {
	if(_d_cells != NULL) {
		CUDA_SAFE_CALL( hipFree(_d_cells) );
		CUDA_SAFE_CALL( hipFree(_d_counters_cells) );
		CUDA_SAFE_CALL( hipFree(_d_matrix_neighs) );
		CUDA_SAFE_CALL( hipFree(_d_number_neighs) );
		CUDA_SAFE_CALL( hipHostFree(_d_cell_overflow) );
	}

	if(this->_use_edge) {
		CUDA_SAFE_CALL( hipFree(_d_edge_list) );
		CUDA_SAFE_CALL( hipFree(_d_number_neighs_no_doubles) );
	}
}

template<typename number, typename number4>
void CUDASimpleVerletList<number, number4>::get_settings(input_file &inp) {
	getInputBool(&inp, "cells_auto_optimisation", &_auto_optimisation, 0);
	getInputNumber(&inp, "verlet_skin", &_verlet_skin, 1);
	getInputNumber(&inp, "max_density_multiplier", &_max_density_multiplier, 0);
	getInputBool(&inp, "use_edge", &this->_use_edge, 0);
	if(this->_use_edge) OX_LOG(Logger::LOG_INFO, "Using edge-based approach...");
}

template<typename number, typename number4>
void CUDASimpleVerletList<number, number4>::_init_cells() {
	number4 box_sides_n4 = this->_h_cuda_box->box_sides();
	number box_sides[3] = { box_sides_n4.x, box_sides_n4.y, box_sides_n4.z };
	number max_factor = pow(2.*this->_N/this->_h_cuda_box->V(), 1./3.);

	for(int i = 0; i < 3; i++) {
		_N_cells_side[i] = (int) (floor(box_sides[i] / sqrt(_sqr_rverlet)) + 0.1);
		if(_N_cells_side[i] < 3) _N_cells_side[i] = 3;
		if(_auto_optimisation && _N_cells_side[i] > ceil(max_factor*box_sides[i])) _N_cells_side[i] = ceil(max_factor*box_sides[i]);
	}

	_N_cells = _N_cells_side[0]*_N_cells_side[1]*_N_cells_side[2];

	number V_cell = this->_h_cuda_box->V()/(number)_N_cells;
	number density = this->_N/this->_h_cuda_box->V();
	if(density < 0.1) density = 0.1;
	number density_factor = density*5.*_max_density_multiplier;
	_max_N_per_cell = (int) (V_cell*density_factor);
	if(_max_N_per_cell > this->_N) _max_N_per_cell = this->_N;
	if(_max_N_per_cell < 5) _max_N_per_cell = 5;

	if(_old_N_cells != -1 && _N_cells != _old_N_cells) {
		CUDA_SAFE_CALL( hipFree(_d_cells) );
		CUDA_SAFE_CALL( hipFree(_d_counters_cells) );
		_d_cells = _d_counters_cells = NULL;
		OX_DEBUG("Re-allocating cells on GPU, from %d to %d\n", _old_N_cells, _N_cells);
	}

	if(_d_cells == NULL) {
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_counters_cells, (size_t) _N_cells*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_cells, (size_t) _N_cells*_max_N_per_cell*sizeof(int)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(verlet_N_cells_side), this->_N_cells_side, 3*sizeof(int)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(verlet_max_N_per_cell), &this->_max_N_per_cell, sizeof(int)) );
	}

	_old_N_cells = _N_cells;
}

template<typename number, typename number4>
void CUDASimpleVerletList<number, number4>::init(int N, number rcut, CUDABox<number, number4> *h_cuda_box, CUDABox<number, number4> *d_cuda_box) {
	CUDABaseList<number, number4>::init(N, rcut, h_cuda_box, d_cuda_box);

	number rverlet = rcut + 2*_verlet_skin;
	_sqr_rverlet = SQR(rverlet);
	_sqr_verlet_skin = SQR(_verlet_skin);
	_vec_size = N*sizeof(number4);

	// volume of a sphere whose radius is ceil(rverlet) times the maximum density (sqrt(2)).
	number density = N/h_cuda_box->V();
	if(density < 0.1) density = 0.1;
	number density_factor = density*5.*_max_density_multiplier;
	_max_neigh = (int) ((4*M_PI*pow(ceil(rverlet), 3) / 3.)*density_factor);
	if(_max_neigh >= N) _max_neigh = N-1;

	_init_cells();

	OX_LOG(Logger::LOG_INFO, "CUDA max_neigh: %d, max_N_per_cell: %d, N_cells: %d (per side: %d %d %d)", _max_neigh, _max_N_per_cell, _N_cells, _N_cells_side[0], _N_cells_side[1], _N_cells_side[2]);
	OX_LOG(Logger::LOG_INFO, "CUDA Cells mem: %.2lf MBs, lists mem: %.2lf MBs", (double) _N_cells*(1 + _max_N_per_cell) * sizeof(int)/1048576., (double) this->_N * (1 + _max_neigh) * sizeof(int)/1048576.);

	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_number_neighs, (size_t) this->_N*sizeof(int)) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_matrix_neighs, (size_t) this->_N*_max_neigh * sizeof(int)) );

	CUDA_SAFE_CALL( hipHostMalloc(&_d_cell_overflow, sizeof(bool), hipHostMallocDefault) );
	_d_cell_overflow[0] = false;

	if(this->_use_edge) {
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_edge_list, (size_t) this->_N*_max_neigh*sizeof(edge_bond)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc(&_d_number_neighs_no_doubles, (size_t) (this->_N + 1)*sizeof(int)) );
	}

	if(_cells_kernel_cfg.threads_per_block == 0) _cells_kernel_cfg.threads_per_block = 64;
	_cells_kernel_cfg.blocks.x = this->_N / _cells_kernel_cfg.threads_per_block + ((this->_N % _cells_kernel_cfg.threads_per_block == 0) ? 0 : 1);
	_cells_kernel_cfg.blocks.y = _cells_kernel_cfg.blocks.z = 1;

	OX_DEBUG("Cells kernel cfg: threads_per_block = %d, blocks = (%d, %d, %d)", _cells_kernel_cfg.threads_per_block,
			_cells_kernel_cfg.blocks.x, _cells_kernel_cfg.blocks.y, _cells_kernel_cfg.blocks.z);

	float f_copy = this->_sqr_rverlet;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(verlet_sqr_rverlet), &f_copy, sizeof(float)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(verlet_N), &this->_N, sizeof(int)) );
}

template<typename number, typename number4>
void CUDASimpleVerletList<number, number4>::update(number4 *poss, number4 *list_poss, LR_bonds *bonds) {
	_init_cells();
	CUDA_SAFE_CALL( hipMemset(_d_counters_cells, 0, _N_cells*sizeof(int)) );

	// fill cells
	simple_fill_cells<number, number4>
		<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
		(poss, _d_cells, _d_counters_cells, _d_cell_overflow, this->_d_cuda_box);
	CUT_CHECK_ERROR("fill_cells (SimpleVerlet) error");

	hipDeviceSynchronize();
	if(_d_cell_overflow[0] == true) throw oxDNAException("A cell contains more than _max_n_per_cell (%d) particles. Please increase the value of max_density_multiplier (which defaults to 1) in the input file\n", _max_N_per_cell);

	// texture binding for the number of particles contained in each cell
	hipBindTexture(0, counters_cells_tex, _d_counters_cells, sizeof(int)*_N_cells);

	// for edge based approach
	if(this->_use_edge) {
		edge_update_neigh_list<number, number4>
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(poss, list_poss, _d_cells, _d_matrix_neighs, _d_number_neighs, _d_number_neighs_no_doubles, bonds, this->_d_cuda_box);
		CUT_CHECK_ERROR("edge_update_neigh_list (SimpleVerlet) error");

		// thrust operates on the GPU
		thrust::device_ptr<int> _d_number_neighs_no_doubles_w (_d_number_neighs_no_doubles);
		_d_number_neighs_no_doubles_w[this->_N] = 0;
		thrust::exclusive_scan(_d_number_neighs_no_doubles_w, _d_number_neighs_no_doubles_w + this->_N + 1, _d_number_neighs_no_doubles_w);
		_N_edges = _d_number_neighs_no_doubles_w[this->_N];
		// get edge list from matrix_neighs
		compress_matrix_neighs
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(_d_matrix_neighs, _d_number_neighs, _d_number_neighs_no_doubles, _d_edge_list);
		CUT_CHECK_ERROR("compress_matrix_neighs error");
	}
	else {
		simple_update_neigh_list<number, number4>
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(poss, list_poss, _d_cells, _d_matrix_neighs, _d_number_neighs, bonds, this->_d_cuda_box);
		CUT_CHECK_ERROR("update_neigh_list (SimpleVerlet) error");
	}

	hipUnbindTexture(counters_cells_tex);
}

template class CUDASimpleVerletList<float, float4>;
template class CUDASimpleVerletList<double, LR_double4>;
